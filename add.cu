#include "hip/hip_runtime.h"
// Simple ADD kernel to demonstrate the general pattern in C CUDA
// compile: nvcc -o add add.cu
#include "data_generator.h"
#include <hip/hip_runtime.h>
#include <iostream>
#define COUNT 100000

// KERNEL
__global__ void MainCUDAKernel(int *a, int *b) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  a[id] *= b[id] + (a[id] % b[id]);
}

int main() {
  srand(time(NULL));
  // GENERATE & ALLOCATE DATA ON HOST
  int dim = COUNT;
  int size = sizeof(int) * dim;
  int h_a[COUNT];
  int h_b[COUNT];
  populate(h_a, dim, false);
  populate(h_b, dim, false);

  // ALLOCATE DATA ON DEVICE AND COPY
  int *d_a, *d_b;
  if (hipMalloc(&d_a, size) != hipSuccess) {
    std::cerr << "Failed: hipMalloc d_a" << std::endl;
    return -1;
  };
  if (hipMalloc(&d_b, size) != hipSuccess) {
    hipFree(d_a);
    std::cerr << "Failed: hipMalloc d_b" << std::endl;
    return -1;
  };
  if (hipMemcpy(d_a, &h_a, size, hipMemcpyHostToDevice) != hipSuccess) {
    hipFree(d_a);
    hipFree(d_b);
    std::cerr << "Failed: hipMemcpy h_a" << std::endl;
    return -1;
  };
  if (hipMemcpy(d_b, &h_b, size, hipMemcpyHostToDevice) != hipSuccess) {
    hipFree(d_a);
    hipFree(d_b);
    std::cerr << "Failed: hipMemcpy h_b" << std::endl;
    return -1;
  };

  // CALL KERNEL
  //   dim (number of threads) must be < 1024
  MainCUDAKernel<<<(dim / 1024) + 1, 1024>>>(d_a, d_b);

  //   OUTPUT
  if (hipMemcpy(&h_a, d_a, size, hipMemcpyDeviceToHost) != hipSuccess) {
    hipFree(d_a);
    hipFree(d_b);
    std::cerr << "Failed: hipMemcpy d_a " << std::endl;
    return -1;
  };
  //   for (int i = 0; i < dim; i++)
  //     std::cout << "a[" << i << "] = " << h_a[i] << std::endl;

  //  CLEANUP
  hipFree(d_a);
  hipFree(d_b);

  return 0;
}